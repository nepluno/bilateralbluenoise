#include <thrust/device_vector.h>
#include <thrust/sort.h>

template <class T>
void thrust_stable_sort(T* ptr, size_t num) {
	thrust::device_ptr<T> thrust_ptr(ptr);
	thrust::stable_sort(thrust_ptr, thrust_ptr + num);
}

extern void thrust_stable_sort(unsigned __int64* ptr, size_t num) {
	thrust_stable_sort<unsigned __int64>(ptr, num);
}